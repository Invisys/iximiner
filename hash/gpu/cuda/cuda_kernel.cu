#include "hip/hip_runtime.h"
#include <hip/driver_types.h>

#include "../../../common/common.h"
#include "../../../app/arguments.h"

#include "../../hasher.h"
#include "../../argon2/argon2.h"

#include "cuda_hasher.h"

#define ITEMS_PER_SEGMENT               32
#define BLOCK_SIZE_UINT4                64
#define BLOCK_SIZE_UINT                256
#define KERNEL_WORKGROUP_SIZE   		32
#define ARGON2_PREHASH_DIGEST_LENGTH_UINT   16
#define ARGON2_PREHASH_SEED_LENGTH_UINT     18
#define IXIAN_SEED_SIZE_UINT                39


#include "blake2b.cu"

#define COMPUTE	\
	asm ("{"	\
		".reg .u32 s1, s2, s3, s4;\n\t"	\
		"mul.lo.u32 s3, %0, %2;\n\t"	\
		"mul.hi.u32 s4, %0, %2;\n\t"	\
		"add.cc.u32 s3, s3, s3;\n\t"	\
		"addc.u32 s4, s4, s4;\n\t"	\
		"add.cc.u32 s1, %0, %2;\n\t"	\
		"addc.u32 s2, %1, %3;\n\t"	\
		"add.cc.u32 %0, s1, s3;\n\t"	\
		"addc.u32 %1, s2, s4;\n\t"	\
		"xor.b32 s1, %0, %6;\n\t"	\
		"xor.b32 %6, %1, %7;\n\t"	\
		"mov.b32 %7, s1;\n\t"	\
		"mul.lo.u32 s3, %4, %6;\n\t"	\
		"mul.hi.u32 s4, %4, %6;\n\t"	\
		"add.cc.u32 s3, s3, s3;\n\t"	\
		"addc.u32 s4, s4, s4;\n\t"	\
		"add.cc.u32 s1, %4, %6;\n\t"	\
		"addc.u32 s2, %5, %7;\n\t"	\
		"add.cc.u32 %4, s1, s3;\n\t"	\
		"addc.u32 %5, s2, s4;\n\t"	\
		"xor.b32 s3, %2, %4;\n\t"	\
		"xor.b32 s4, %3, %5;\n\t"	\
		"shf.r.wrap.b32 %3, s4, s3, 24;\n\t"	\
		"shf.r.wrap.b32 %2, s3, s4, 24;\n\t"	\
		"mul.lo.u32 s3, %0, %2;\n\t"	\
		"mul.hi.u32 s4, %0, %2;\n\t"	\
		"add.cc.u32 s3, s3, s3;\n\t"	\
		"addc.u32 s4, s4, s4;\n\t"	\
		"add.cc.u32 s1, %0, %2;\n\t"	\
		"addc.u32 s2, %1, %3;\n\t"	\
		"add.cc.u32 %0, s1, s3;\n\t"	\
		"addc.u32 %1, s2, s4;\n\t"	\
		"xor.b32 s3, %0, %6;\n\t"	\
		"xor.b32 s4, %1, %7;\n\t"	\
		"shf.r.wrap.b32 %7, s4, s3, 16;\n\t"	\
		"shf.r.wrap.b32 %6, s3, s4, 16;\n\t"	\
		"mul.lo.u32 s3, %4, %6;\n\t"	\
		"mul.hi.u32 s4, %4, %6;\n\t"	\
		"add.cc.u32 s3, s3, s3;\n\t"	\
		"addc.u32 s4, s4, s4;\n\t"	\
		"add.cc.u32 s1, %4, %6;\n\t"	\
		"addc.u32 s2, %5, %7;\n\t"	\
		"add.cc.u32 %4, s1, s3;\n\t"	\
		"addc.u32 %5, s2, s4;\n\t"	\
		"xor.b32 s3, %2, %4;\n\t"	\
		"xor.b32 s4, %3, %5;\n\t"	\
		"shf.r.wrap.b32 %3, s3, s4, 31;\n\t"	\
		"shf.r.wrap.b32 %2, s4, s3, 31;\n\t"	\
	"}" : "+r"(tmp_a.x), "+r"(tmp_a.y), "+r"(tmp_a.z), "+r"(tmp_a.w), "+r"(tmp_b.x), "+r"(tmp_b.y), "+r"(tmp_b.z), "+r"(tmp_b.w));

#define G1(data)           \
{                           \
	COMPUTE \
	tmp_a.z = __shfl_sync(0xffffffff, tmp_a.z, i_shfl1_1); \
	tmp_a.w = __shfl_sync(0xffffffff, tmp_a.w, i_shfl1_1); \
	tmp_b.x = __shfl_sync(0xffffffff, tmp_b.x, i_shfl1_2); \
	tmp_b.y = __shfl_sync(0xffffffff, tmp_b.y, i_shfl1_2); \
	tmp_b.z = __shfl_sync(0xffffffff, tmp_b.z, i_shfl1_3); \
	tmp_b.w = __shfl_sync(0xffffffff, tmp_b.w, i_shfl1_3); \
}

#define G2(data)           \
{ \
	COMPUTE \
    data[i2_0_0] = tmp_a.x; \
    data[i2_0_1] = tmp_a.y; \
    data[i2_1_0] = tmp_a.z; \
    data[i2_1_1] = tmp_a.w; \
    data[i2_2_0] = tmp_b.x; \
    data[i2_2_1] = tmp_b.y; \
    data[i2_3_0] = tmp_b.z; \
    data[i2_3_1] = tmp_b.w; \
    __syncwarp(); \
}

#define G3(data)           \
{                           \
    tmp_a.x = data[i3_0_0]; \
    tmp_a.y = data[i3_0_1]; \
    tmp_a.z = data[i3_1_0]; \
    tmp_a.w = data[i3_1_1]; \
    tmp_b.x = data[i3_2_0]; \
    tmp_b.y = data[i3_2_1]; \
    tmp_b.z = data[i3_3_0]; \
    tmp_b.w = data[i3_3_1]; \
	COMPUTE \
	tmp_a.z = __shfl_sync(0xffffffff, tmp_a.z, i_shfl2_1); \
	tmp_a.w = __shfl_sync(0xffffffff, tmp_a.w, i_shfl2_1); \
	tmp_b.x = __shfl_sync(0xffffffff, tmp_b.x, i_shfl2_2); \
	tmp_b.y = __shfl_sync(0xffffffff, tmp_b.y, i_shfl2_2); \
	tmp_b.z = __shfl_sync(0xffffffff, tmp_b.z, i_shfl2_3); \
	tmp_b.w = __shfl_sync(0xffffffff, tmp_b.w, i_shfl2_3); \
}

#define G4(data)           \
{                           \
	COMPUTE \
    data[i4_0_0] = tmp_a.x; \
    data[i4_0_1] = tmp_a.y; \
    data[i4_1_0] = tmp_a.z; \
    data[i4_1_1] = tmp_a.w; \
    data[i4_2_0] = tmp_b.x; \
    data[i4_2_1] = tmp_b.y; \
    data[i4_3_0] = tmp_b.z; \
    data[i4_3_1] = tmp_b.w; \
    __syncwarp(); \
    tmp_a.x = data[i1_0_0]; \
    tmp_a.y = data[i1_0_1]; \
    tmp_a.z = data[i1_1_0]; \
    tmp_a.w = data[i1_1_1]; \
    tmp_b.x = data[i1_2_0]; \
    tmp_b.y = data[i1_2_1]; \
    tmp_b.z = data[i1_3_0]; \
    tmp_b.w = data[i1_3_1]; \
}

__constant__ int offsets[768] = {
		0, 4, 8, 12,
		1, 5, 9, 13,
		2, 6, 10, 14,
		3, 7, 11, 15,
		16, 20, 24, 28,
		17, 21, 25, 29,
		18, 22, 26, 30,
		19, 23, 27, 31,
		32, 36, 40, 44,
		33, 37, 41, 45,
		34, 38, 42, 46,
		35, 39, 43, 47,
		48, 52, 56, 60,
		49, 53, 57, 61,
		50, 54, 58, 62,
		51, 55, 59, 63,
		64, 68, 72, 76,
		65, 69, 73, 77,
		66, 70, 74, 78,
		67, 71, 75, 79,
		80, 84, 88, 92,
		81, 85, 89, 93,
		82, 86, 90, 94,
		83, 87, 91, 95,
		96, 100, 104, 108,
		97, 101, 105, 109,
		98, 102, 106, 110,
		99, 103, 107, 111,
		112, 116, 120, 124,
		113, 117, 121, 125,
		114, 118, 122, 126,
		115, 119, 123, 127,
		0, 5, 10, 15,
		1, 6, 11, 12,
		2, 7, 8, 13,
		3, 4, 9, 14,
		16, 21, 26, 31,
		17, 22, 27, 28,
		18, 23, 24, 29,
		19, 20, 25, 30,
		32, 37, 42, 47,
		33, 38, 43, 44,
		34, 39, 40, 45,
		35, 36, 41, 46,
		48, 53, 58, 63,
		49, 54, 59, 60,
		50, 55, 56, 61,
		51, 52, 57, 62,
		64, 69, 74, 79,
		65, 70, 75, 76,
		66, 71, 72, 77,
		67, 68, 73, 78,
		80, 85, 90, 95,
		81, 86, 91, 92,
		82, 87, 88, 93,
		83, 84, 89, 94,
		96, 101, 106, 111,
		97, 102, 107, 108,
		98, 103, 104, 109,
		99, 100, 105, 110,
		112, 117, 122, 127,
		113, 118, 123, 124,
		114, 119, 120, 125,
		115, 116, 121, 126,
		0, 32, 64, 96,
		1, 33, 65, 97,
		2, 34, 66, 98,
		3, 35, 67, 99,
		4, 36, 68, 100,
		5, 37, 69, 101,
		6, 38, 70, 102,
		7, 39, 71, 103,
		8, 40, 72, 104,
		9, 41, 73, 105,
		10, 42, 74, 106,
		11, 43, 75, 107,
		12, 44, 76, 108,
		13, 45, 77, 109,
		14, 46, 78, 110,
		15, 47, 79, 111,
		16, 48, 80, 112,
		17, 49, 81, 113,
		18, 50, 82, 114,
		19, 51, 83, 115,
		20, 52, 84, 116,
		21, 53, 85, 117,
		22, 54, 86, 118,
		23, 55, 87, 119,
		24, 56, 88, 120,
		25, 57, 89, 121,
		26, 58, 90, 122,
		27, 59, 91, 123,
		28, 60, 92, 124,
		29, 61, 93, 125,
		30, 62, 94, 126,
		31, 63, 95, 127,
		0, 33, 80, 113,
		1, 48, 81, 96,
		2, 35, 82, 115,
		3, 50, 83, 98,
		4, 37, 84, 117,
		5, 52, 85, 100,
		6, 39, 86, 119,
		7, 54, 87, 102,
		8, 41, 88, 121,
		9, 56, 89, 104,
		10, 43, 90, 123,
		11, 58, 91, 106,
		12, 45, 92, 125,
		13, 60, 93, 108,
		14, 47, 94, 127,
		15, 62, 95, 110,
		16, 49, 64, 97,
		17, 32, 65, 112,
		18, 51, 66, 99,
		19, 34, 67, 114,
		20, 53, 68, 101,
		21, 36, 69, 116,
		22, 55, 70, 103,
		23, 38, 71, 118,
		24, 57, 72, 105,
		25, 40, 73, 120,
		26, 59, 74, 107,
		27, 42, 75, 122,
		28, 61, 76, 109,
		29, 44, 77, 124,
		30, 63, 78, 111,
		31, 46, 79, 126,
        0, 1, 2, 3,
        1, 2, 3, 0,
        2, 3, 0, 1,
        3, 0, 1, 2,
        4, 5, 6, 7,
        5, 6, 7, 4,
        6, 7, 4, 5,
        7, 4, 5, 6,
        8, 9, 10, 11,
        9, 10, 11, 8,
        10, 11, 8, 9,
        11, 8, 9, 10,
        12, 13, 14, 15,
        13, 14, 15, 12,
        14, 15, 12, 13,
        15, 12, 13, 14,
        16, 17, 18, 19,
        17, 18, 19, 16,
        18, 19, 16, 17,
        19, 16, 17, 18,
        20, 21, 22, 23,
        21, 22, 23, 20,
        22, 23, 20, 21,
        23, 20, 21, 22,
        24, 25, 26, 27,
        25, 26, 27, 24,
        26, 27, 24, 25,
        27, 24, 25, 26,
        28, 29, 30, 31,
        29, 30, 31, 28,
        30, 31, 28, 29,
        31, 28, 29, 30,
        0, 1, 16, 17,
        1, 16, 17, 0,
        2, 3, 18, 19,
        3, 18, 19, 2,
        4, 5, 20, 21,
        5, 20, 21, 4,
        6, 7, 22, 23,
        7, 22, 23, 6,
        8, 9, 24, 25,
        9, 24, 25, 8,
        10, 11, 26, 27,
        11, 26, 27, 10,
        12, 13, 28, 29,
        13, 28, 29, 12,
        14, 15, 30, 31,
        15, 30, 31, 14,
        16, 17, 0, 1,
        17, 0, 1, 16,
        18, 19, 2, 3,
        19, 2, 3, 18,
        20, 21, 4, 5,
        21, 4, 5, 20,
        22, 23, 6, 7,
        23, 6, 7, 22,
        24, 25, 8, 9,
        25, 8, 9, 24,
        26, 27, 10, 11,
        27, 10, 11, 26,
        28, 29, 12, 13,
        29, 12, 13, 28,
        30, 31, 14, 15,
        31, 14, 15, 30
};

inline __host__ __device__ void operator^=( uint4& a, uint4 s) {
   a.x ^= s.x; a.y ^= s.y; a.z ^= s.z; a.w ^= s.w;
}

__global__ void fill_blocks(uint32_t *scratchpad0,
							uint32_t *scratchpad1,
							uint32_t *scratchpad2,
							uint32_t *scratchpad3,
							uint32_t *scratchpad4,
							uint32_t *scratchpad5,
							uint32_t *seed,
							uint32_t *out,
							uint32_t *addresses,
							uint32_t *segments,
							int memsize,
							int threads_per_chunk,
							int thread_idx) {
	__shared__ uint32_t state[2 * BLOCK_SIZE_UINT];
	__shared__ uint32_t addr[2 * 32];

	uint4 tmp_a, tmp_b, tmp_c, tmp_d, tmp_p, tmp_q;

	int hash = blockIdx.x;
	int mem_hash = hash + thread_idx;
	int local_id = threadIdx.x;

	int id = local_id % ITEMS_PER_SEGMENT;
	int segment = local_id / ITEMS_PER_SEGMENT;

	int offset = id << 2;

	int i1_0_0 = 2 * offsets[offset];
	int i1_0_1 = i1_0_0 + 1;
	int i1_1_0 = 2 * offsets[offset + 1];
	int i1_1_1 = i1_1_0 + 1;
	int i1_2_0 = 2 * offsets[offset + 2];
	int i1_2_1 = i1_2_0 + 1;
	int i1_3_0 = 2 * offsets[offset + 3];
	int i1_3_1 = i1_3_0 + 1;

	int i2_0_0 = 2 * offsets[offset + 128];
	int i2_0_1 = i2_0_0 + 1;
	int i2_1_0 = 2 * offsets[offset + 129];
	int i2_1_1 = i2_1_0 + 1;
	int i2_2_0 = 2 * offsets[offset + 130];
	int i2_2_1 = i2_2_0 + 1;
	int i2_3_0 = 2 * offsets[offset + 131];
	int i2_3_1 = i2_3_0 + 1;

	int i3_0_0 = 2 * offsets[offset + 256];
	int i3_0_1 = i3_0_0 + 1;
	int i3_1_0 = 2 * offsets[offset + 257];
	int i3_1_1 = i3_1_0 + 1;
	int i3_2_0 = 2 * offsets[offset + 258];
	int i3_2_1 = i3_2_0 + 1;
	int i3_3_0 = 2 * offsets[offset + 259];
	int i3_3_1 = i3_3_0 + 1;

	int i4_0_0 = 2 * offsets[offset + 384];
	int i4_0_1 = i4_0_0 + 1;
	int i4_1_0 = 2 * offsets[offset + 385];
	int i4_1_1 = i4_1_0 + 1;
	int i4_2_0 = 2 * offsets[offset + 386];
	int i4_2_1 = i4_2_0 + 1;
	int i4_3_0 = 2 * offsets[offset + 387];
	int i4_3_1 = i4_3_0 + 1;

	int i_shfl1_1 = offsets[offset + 513];
	int i_shfl1_2 = offsets[offset + 514];
	int i_shfl1_3 = offsets[offset + 515];
	int i_shfl2_1 = offsets[offset + 641];
	int i_shfl2_2 = offsets[offset + 642];
	int i_shfl2_3 = offsets[offset + 643];

    int scratchpad_location = mem_hash / threads_per_chunk;
    uint4 *memory = reinterpret_cast<uint4*>(scratchpad0);
    if(scratchpad_location == 1) memory = reinterpret_cast<uint4*>(scratchpad1);
    if(scratchpad_location == 2) memory = reinterpret_cast<uint4*>(scratchpad2);
    if(scratchpad_location == 3) memory = reinterpret_cast<uint4*>(scratchpad3);
    if(scratchpad_location == 4) memory = reinterpret_cast<uint4*>(scratchpad4);
    if(scratchpad_location == 5) memory = reinterpret_cast<uint4*>(scratchpad5);
    int hash_offset = mem_hash - scratchpad_location * threads_per_chunk;
    memory = memory + hash_offset * (memsize >> 4);

	uint32_t *mem_seed = seed + hash * 4 * BLOCK_SIZE_UINT;

	uint32_t *seed_src = mem_seed + segment * 2 * BLOCK_SIZE_UINT;
	uint4 *seed_dst = memory + segment * 512 * BLOCK_SIZE_UINT4;

	seed_dst[id] = make_uint4(seed_src[i1_0_0], seed_src[i1_0_1], seed_src[i1_1_0], seed_src[i1_1_1]);
	seed_dst[id + 32] = make_uint4(seed_src[i1_2_0], seed_src[i1_2_1], seed_src[i1_3_0], seed_src[i1_3_1]);
	seed_src += BLOCK_SIZE_UINT;
	seed_dst += BLOCK_SIZE_UINT4;
	seed_dst[id] = make_uint4(seed_src[i1_0_0], seed_src[i1_0_1], seed_src[i1_1_0], seed_src[i1_1_1]);
	seed_dst[id + 32] = make_uint4(seed_src[i1_2_0], seed_src[i1_2_1], seed_src[i1_3_0], seed_src[i1_3_1]);

	uint4 *next_block;
	uint4 *prev_block;
	uint4 *ref_block;

	uint32_t *local_state = state + segment * BLOCK_SIZE_UINT;
	uint32_t *local_addr = addr + segment * 32;

	segments += segment;
	uint16_t addr_start_idx = 0;
	uint16_t prev_blk_idx;
	int inc = 126;

	for(int s=0; s<4; s++) {
		int idx = ((s == 0) ? 2 : 0); // index for first slice in each lane is 2
		uint32_t curr_seg = segments[s * 2];

		asm("mov.b32 {%0, %1}, %2;"
		: "=h"(addr_start_idx), "=h"(prev_blk_idx) : "r"(curr_seg));

		uint32_t *addr = addresses + addr_start_idx;
		uint32_t *stop_addr = addresses + addr_start_idx + inc;
		inc = 128;

		prev_block = memory + prev_blk_idx * BLOCK_SIZE_UINT4;

		tmp_a = prev_block[id];
		tmp_b = prev_block[id + 32];

		__syncthreads();

		for(; addr < stop_addr; addr += 32) {
			local_addr[id] = addr[id];

			uint64_t i_limit = stop_addr - addr;
			if(i_limit > 32) i_limit = 32;

			int16_t addr0, addr1;
			asm("{mov.b32 {%0, %1}, %2;}": "=h"(addr0), "=h"(addr1) : "r"(local_addr[0]));

			if(addr1 != -1) {
				ref_block = memory + addr1 * BLOCK_SIZE_UINT4;
				tmp_p = ref_block[id];
				tmp_q = ref_block[id + 32];
			}

			for(int i=0;i<i_limit;i++, idx++) {
				next_block = memory + addr0 * BLOCK_SIZE_UINT4;

				if(addr1 != -1) {
					tmp_a ^= tmp_p;
					tmp_b ^= tmp_q;

					if (i < (i_limit - 1)) {
						asm("{mov.b32 {%0, %1}, %2;}": "=h"(addr0), "=h"(addr1) : "r"(local_addr[i + 1]));
						ref_block = memory + addr1 * BLOCK_SIZE_UINT4;
						tmp_p = ref_block[id];
						tmp_q = ref_block[id + 32];
					}
				}
				else {
					uint32_t pseudo_rand_lo = __shfl_sync(0xffffffff, tmp_a.x, 0);
					uint32_t pseudo_rand_hi = __shfl_sync(0xffffffff, tmp_a.y, 0);

					uint64_t ref_lane = pseudo_rand_hi % 2; // thr_cost
					uint32_t reference_area_size = 0;
					if (segment == ref_lane) {
						reference_area_size = s * 128 + idx - 1; // seg_length
					} else {
						reference_area_size = s * 128 + ((idx == 0) ? (-1) : 0);
					}
					asm("{mul.hi.u32 %0, %1, %1; mul.hi.u32 %0, %0, %2; }": "=r"(pseudo_rand_lo) : "r"(pseudo_rand_lo), "r"(reference_area_size));

					uint32_t relative_position = reference_area_size - 1 - pseudo_rand_lo;

					addr1 = ref_lane * 512 + relative_position % 512; // lane_length

					ref_block = memory + addr1 * BLOCK_SIZE_UINT4;
					tmp_a ^= ref_block[id];
					tmp_b ^= ref_block[id + 32];

					if (i < (i_limit - 1)) {
						asm("{mov.b32 {%0, %1}, %2;}": "=h"(addr0), "=h"(addr1) : "r"(local_addr[i + 1]));
					}
				}

				tmp_c = tmp_a; tmp_d = tmp_b;

				G1(local_state);
				G2(local_state);
				G3(local_state);
				G4(local_state);

				tmp_a ^= tmp_c; tmp_b ^= tmp_d;

				next_block[id] = tmp_a;
				next_block[id + 32] = tmp_b;
			}
		}
	}

	__syncthreads();

	int dst_addr = 1020;

	uint4 *block = memory + ((int16_t*)(&addresses[dst_addr]))[0] * BLOCK_SIZE_UINT4;
	uint4 data = block[id + segment * 32];

	block = memory + ((int16_t*)(&addresses[dst_addr]))[1] * BLOCK_SIZE_UINT4;
	data ^= block[id + segment * 32];

	int idx0 = (segment == 0) ? i1_0_0 : i1_2_0;
	int idx1 = (segment == 0) ? i1_0_1 : i1_2_1;
	int idx2 = (segment == 0) ? i1_1_0 : i1_3_0;
	int idx3 = (segment == 0) ? i1_1_1 : i1_3_1;

	uint32_t *out_mem = out + hash * BLOCK_SIZE_UINT;
	out_mem[idx0] = data.x;
	out_mem[idx1] = data.y;
	out_mem[idx2] = data.z;
	out_mem[idx3] = data.w;
};

__global__ void prehash (
        uint32_t *preseed,
        uint32_t *seed) {
    extern __shared__ uint32_t shared[]; // size = 8 * 88

    int hash = blockIdx.x * 2;
    int id = threadIdx.x; // 32 threads
    int hash_idx = id >> 4;
    hash += hash_idx;
    id = id & 0xF;

    int thr_id = id % 4; // thread id in session
    int session = id / 4; // 4 blake2b hashing session
    int lane = session / 2;  // 2 lanes
    int idx = session % 2; // idx in lane

    uint32_t *local_mem = &shared[(hash_idx * 4 + session) * BLAKE_SHARED_MEM_UINT];
    uint32_t *local_preseed = preseed + hash * IXIAN_SEED_SIZE_UINT;
    uint32_t *local_seed = seed + (hash * 4 + session) * BLOCK_SIZE_UINT;

    uint64_t *h = (uint64_t*)&local_mem[20];
    uint32_t *buf = (uint32_t*)&h[10];
    uint32_t *value = &buf[32];

    int buf_len = blake2b_init(h, ARGON2_PREHASH_DIGEST_LENGTH_UINT, thr_id);
    *value = 2; //lanes
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    *value = 32; //outlen
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    *value = 1024; //m_cost
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    *value = 1; //t_cost
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    *value = ARGON2_VERSION; //version
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    *value = ARGON2_TYPE_VALUE; //type
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    *value = 92; //pw_len
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    buf_len = blake2b_update(local_preseed, 23, h, buf, buf_len, thr_id);
    *value = 64; //salt_len
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    buf_len = blake2b_update(local_preseed + 23, 16, h, buf, buf_len, thr_id);
    *value = 0; //secret_len
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    buf_len = blake2b_update(NULL, 0, h, buf, buf_len, thr_id);
    *value = 0; //ad_len
    buf_len = blake2b_update(value, 1, h, buf, buf_len, thr_id);
    buf_len = blake2b_update(NULL, 0, h, buf, buf_len, thr_id);

    blake2b_final(local_mem, ARGON2_PREHASH_DIGEST_LENGTH_UINT, h, buf, buf_len, thr_id);

    if (thr_id == 0) {
        local_mem[ARGON2_PREHASH_DIGEST_LENGTH_UINT] = idx;
        local_mem[ARGON2_PREHASH_DIGEST_LENGTH_UINT + 1] = lane;
    }

    blake2b_digestLong(local_seed, ARGON2_DWORDS_IN_BLOCK, local_mem, ARGON2_PREHASH_SEED_LENGTH_UINT, thr_id, &local_mem[20]);
}

__global__ void posthash (
        uint32_t *hash,
        uint32_t *out) {
    extern __shared__ uint32_t shared[]; // size = 88

    int hash_id = blockIdx.x;
    int thread = threadIdx.x;

    int thr_id = thread % 4;
    int session = thread / 4;

    uint32_t *local_mem = &shared[session * BLAKE_SHARED_MEM_UINT];
    uint32_t *local_hash = hash + (hash_id * 8 + session) * ARGON2_RAW_LENGTH / 4;
    uint32_t *local_out = out + (hash_id * 8 + session) * BLOCK_SIZE_UINT;

    blake2b_digestLong(local_hash, ARGON2_RAW_LENGTH / 4, local_out, ARGON2_DWORDS_IN_BLOCK, thr_id, local_mem);
}

void cuda_allocate(cuda_device_info *device, double chunks, size_t chunk_size) {
	device->error = hipSetDevice(device->cuda_index);
	if(device->error != hipSuccess) {
		device->error_message = "Error setting current device for memory allocation.";
		return;
	}

	size_t allocated_mem_for_current_chunk = 0;

	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memory_chunk_0, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memory_chunk_1, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memory_chunk_2, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memory_chunk_3, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memory_chunk_4, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	if (chunks > 0) {
		allocated_mem_for_current_chunk = chunks > 1 ? chunk_size : (size_t)ceil(chunk_size * chunks);
		chunks -= 1;
	}
	else {
		allocated_mem_for_current_chunk = 1;
	}
	device->error = hipMalloc(&device->arguments.memory_chunk_5, allocated_mem_for_current_chunk);
	if (device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}

	//optimise address sizes
	uint16_t *addresses = (uint16_t *)malloc(argon2profile_default->block_refs_size * 2 * sizeof(uint16_t));
	for(int i=0;i<argon2profile_default->block_refs_size;i++) {
		addresses[i*2] = argon2profile_default->block_refs[i*4 + (i >= 1020 ? 1 : 0)];
		addresses[i*2 + 1] = argon2profile_default->block_refs[i*4 + 2];
		if(argon2profile_default->block_refs[i*4 + 3] == 0) {
			addresses[i*2] |= 32768;
		}
	}
	device->error = hipMalloc(&device->arguments.address, argon2profile_default->block_refs_size * 2 * sizeof(uint16_t));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.address, addresses, argon2profile_default->block_refs_size * 2 * sizeof(uint16_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}
	free(addresses);

	//reorganize segments data
	uint16_t *segments = (uint16_t *)malloc(8 * 2 * sizeof(uint16_t));
	for(int i=0;i<8;i++) {
		int seg_start = argon2profile_default->segments[i*3];
		segments[i*2] = seg_start;
		segments[i*2 + 1] = argon2profile_default->block_refs[seg_start*4 + 1];
	}
	device->error = hipMalloc(&device->arguments.segments, 8 * 2 * sizeof(uint16_t));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.segments, segments, 8 * 2 * sizeof(uint16_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}
	free(segments);

	size_t preseed_memory_size = device->profile_info.threads * IXIAN_SEED_SIZE;
    size_t seed_memory_size = device->profile_info.threads * 4 * ARGON2_BLOCK_SIZE;
    size_t out_memory_size = device->profile_info.threads * ARGON2_BLOCK_SIZE;
    size_t hash_memory_size = device->profile_info.threads * ARGON2_RAW_LENGTH;

    device->error = hipMalloc(&device->arguments.preseed_memory[0], preseed_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.seed_memory[0], seed_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.out_memory[0], out_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.hash_memory[0], hash_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating memory.";
        return;
    }
    device->error = hipHostMalloc(&device->arguments.host_seed_memory[0], preseed_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating pinned memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.preseed_memory[1], preseed_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.seed_memory[1], seed_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.out_memory[1], out_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating memory.";
        return;
    }
    device->error = hipMalloc(&device->arguments.hash_memory[1], hash_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating memory.";
        return;
    }
    device->error = hipHostMalloc(&device->arguments.host_seed_memory[1], preseed_memory_size);
    if (device->error != hipSuccess) {
        device->error_message = "Error allocating pinned memory.";
        return;
    }
}

void cuda_free(cuda_device_info *device) {
	hipSetDevice(device->cuda_index);

	if(device->arguments.address != NULL) {
		hipFree(device->arguments.address);
		device->arguments.address = NULL;
	}

	if(device->arguments.segments != NULL) {
		hipFree(device->arguments.segments);
		device->arguments.segments = NULL;
	}

    if(device->arguments.memory_chunk_0 != NULL) {
        hipFree(device->arguments.memory_chunk_0);
        device->arguments.memory_chunk_0 = NULL;
    }

    if(device->arguments.memory_chunk_1 != NULL) {
        hipFree(device->arguments.memory_chunk_1);
        device->arguments.memory_chunk_1 = NULL;
    }

    if(device->arguments.memory_chunk_2 != NULL) {
        hipFree(device->arguments.memory_chunk_2);
        device->arguments.memory_chunk_2 = NULL;
    }

    if(device->arguments.memory_chunk_3 != NULL) {
        hipFree(device->arguments.memory_chunk_3);
        device->arguments.memory_chunk_3 = NULL;
    }

    if(device->arguments.memory_chunk_4 != NULL) {
        hipFree(device->arguments.memory_chunk_4);
        device->arguments.memory_chunk_4 = NULL;
    }

    if(device->arguments.memory_chunk_5 != NULL) {
        hipFree(device->arguments.memory_chunk_5);
        device->arguments.memory_chunk_5 = NULL;
    }

    if(device->arguments.preseed_memory != NULL) {
        for(int i=0;i<2;i++) {
            if(device->arguments.preseed_memory[i] != NULL)
                hipFree(device->arguments.preseed_memory[i]);
            device->arguments.preseed_memory[i] = NULL;
        }
    }

	if(device->arguments.seed_memory != NULL) {
		for(int i=0;i<2;i++) {
			if(device->arguments.seed_memory[i] != NULL)
				hipFree(device->arguments.seed_memory[i]);
			device->arguments.seed_memory[i] = NULL;
		}
	}

	if(device->arguments.out_memory != NULL) {
		for(int i=0;i<2;i++) {
			if(device->arguments.out_memory[i] != NULL)
				hipFree(device->arguments.out_memory[i]);
			device->arguments.out_memory[i] = NULL;
		}
	}

    if(device->arguments.hash_memory != NULL) {
        for(int i=0;i<2;i++) {
            if(device->arguments.hash_memory[i] != NULL)
                hipFree(device->arguments.hash_memory[i]);
            device->arguments.hash_memory[i] = NULL;
        }
    }

	if(device->arguments.host_seed_memory != NULL) {
		for(int i=0;i<2;i++) {
			if(device->arguments.host_seed_memory[i] != NULL)
				hipHostFree(device->arguments.host_seed_memory[i]);
			device->arguments.host_seed_memory[i] = NULL;
		}
	}

	hipDeviceReset();
}

bool cuda_kernel_prehasher(void *memory, int threads, argon2profile *profile, void *user_data) {
    cuda_gpumgmt_thread_data *gpumgmt_thread = (cuda_gpumgmt_thread_data *)user_data;
    cuda_device_info *device = gpumgmt_thread->device;
    hipStream_t stream = (hipStream_t)gpumgmt_thread->device_data;

    size_t work_items = 32;

    gpumgmt_thread->lock();

    device->error = hipMemcpyAsync(device->arguments.preseed_memory[gpumgmt_thread->thread_id], memory, threads * IXIAN_SEED_SIZE, hipMemcpyHostToDevice, stream);
    if (device->error != hipSuccess) {
        device->error_message = "Error writing to gpu memory.";
        gpumgmt_thread->unlock();
        return false;
    }

    prehash <<<threads / 2, work_items, 8 * BLAKE_SHARED_MEM, stream>>> (
                device->arguments.preseed_memory[gpumgmt_thread->thread_id],
                device->arguments.seed_memory[gpumgmt_thread->thread_id]);

    return true;
}

void *cuda_kernel_filler(void *memory, int threads, argon2profile *profile, void *user_data) {
	cuda_gpumgmt_thread_data *gpumgmt_thread = (cuda_gpumgmt_thread_data *)user_data;
	cuda_device_info *device = gpumgmt_thread->device;
	hipStream_t stream = (hipStream_t)gpumgmt_thread->device_data;

	uint32_t memsize = (uint32_t)argon2profile_default->memsize;
	uint32_t parallelism = argon2profile_default->thr_cost;

    size_t work_items = KERNEL_WORKGROUP_SIZE * parallelism;

	fill_blocks <<<threads, work_items, 0, stream>>> ((uint32_t*)device->arguments.memory_chunk_0,
			(uint32_t*)device->arguments.memory_chunk_1,
			(uint32_t*)device->arguments.memory_chunk_2,
			(uint32_t*)device->arguments.memory_chunk_3,
			(uint32_t*)device->arguments.memory_chunk_4,
			(uint32_t*)device->arguments.memory_chunk_5,
			device->arguments.seed_memory[gpumgmt_thread->thread_id],
			device->arguments.out_memory[gpumgmt_thread->thread_id],
			device->arguments.address,
			device->arguments.segments,
			memsize, device->profile_info.threads_per_chunk, gpumgmt_thread->threads_idx);

	return memory;
}

bool cuda_kernel_posthasher(void *memory, int threads, argon2profile *profile, void *user_data) {
	cuda_gpumgmt_thread_data *gpumgmt_thread = (cuda_gpumgmt_thread_data *)user_data;
	cuda_device_info *device = gpumgmt_thread->device;
	hipStream_t stream = (hipStream_t)gpumgmt_thread->device_data;

    size_t work_items = 32;

	posthash <<<threads / 8, work_items, 8 * BLAKE_SHARED_MEM, stream>>> (
            device->arguments.hash_memory[gpumgmt_thread->thread_id],
            device->arguments.out_memory[gpumgmt_thread->thread_id]);

	device->error = hipMemcpyAsync(memory, device->arguments.hash_memory[gpumgmt_thread->thread_id], threads * ARGON2_RAW_LENGTH, hipMemcpyDeviceToHost, stream);
	if (device->error != hipSuccess) {
		device->error_message = "Error reading gpu memory.";
		gpumgmt_thread->unlock();
		return false;
	}

	while(hipStreamQuery(stream) != hipSuccess) {
		this_thread::sleep_for(chrono::milliseconds(10));
		continue;
	}

	gpumgmt_thread->unlock();


	return memory;
}